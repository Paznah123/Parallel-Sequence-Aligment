#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "header.h"

// ======================== macros for allocating device memory and error check

#define CUDA_ERR_CHECK(err,msg) (\
		{if (err != hipSuccess) { \
			fprintf(stderr, msg " - %s\n", hipGetErrorString(err)); \
			exit(EXIT_FAILURE); \
		} \
	})

#define CUDA_MEM_INIT(dest, src, size, type) {\
	hipError_t err = hipSuccess;\
	size_t  arrSize = size * sizeof(type);\
	err = hipMalloc((void**)&dest, arrSize);\
	CUDA_ERR_CHECK(err, "Failed to allocate device memory");\
	err = hipMemcpy(dest, src, arrSize, hipMemcpyHostToDevice);\
	CUDA_ERR_CHECK(err, "Failed to copy data from host to device"); }\

// ======================== calculate mutant score

__device__ float calcMutantScore(char* seq1, char* seq2, float* ABCgrid, int len2, int n, int k)
{
	float score = 0;
	int i = 0, j = 0;
	
	for (; j < len2; i++, j++)
	{
		if (j == n || j == k) 
			j++;
		int fc_idx = seq1[i] - 'A';
		int sc_idx = seq2[j] - 'A';
		score += ABCgrid[fc_idx*ABC_NUM + sc_idx];
	}	

 	return score;	
}

// ======================== find mutant best score

__global__ void calcMutantBestScoreKernel(char* d_seq1, char* d_seq2, float* d_bestScores, int* d_bestOffsets, 
						int* d_nkArr, float* d_ABCgrid, int num_mutants, int maxOffset, int len2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = 0;
	float bestScore = -10000000000.0;
	
	if (i < num_mutants)
	{
		for (int j = 0; j <= maxOffset; j++)
		{
			float score = calcMutantScore(d_seq1 + j, d_seq2, d_ABCgrid, len2, d_nkArr[i], d_nkArr[i+num_mutants]);
			if (score > bestScore)
			{
				bestScore = score;
				offset = j;
			}
		}
		d_bestScores[i] = bestScore;
		d_bestOffsets[i] = offset;
	}

}

// ======================== entry point to CUDA

void calcBestScoreCUDA(char* seq1, char* seq2, Data data)
{
	int len1 = strlen(seq1);
	int maxOffset = len1 - (data.len2-2);

	char* d_seq1 = NULL; // allocate seq1 memory
	CUDA_MEM_INIT(d_seq1, seq1, len1, char);
	
	char* d_seq2 = NULL; // allocate seq2 memory
	CUDA_MEM_INIT(d_seq2, seq2, data.len2, char);
	
	int* d_nkArr = NULL; // allocate nk array memory
	CUDA_MEM_INIT(d_nkArr, data.nkArr, data.num_mutants*2, int);

	float* d_ABCgrid = NULL; // allocate ABC grid memory
	CUDA_MEM_INIT(d_ABCgrid, data.ABCgrid, ABC_NUM*ABC_NUM, float);
	
	float* d_bestScores = NULL;  // allocate best scores memory
	CUDA_MEM_INIT(d_bestScores, data.bestScores, data.num_mutants, float);

	int* d_bestOffsets = NULL;  // allocate best offsets memory
	CUDA_MEM_INIT(d_bestOffsets, data.bestOffsets, data.num_mutants, int);
	
	int threads = 32;
	int blocks = (data.num_mutants + threads-1) / threads;
	
	calcMutantBestScoreKernel<<<blocks, threads>>>(d_seq1, d_seq2, d_bestScores, d_bestOffsets, d_nkArr, d_ABCgrid, data.num_mutants, maxOffset, data.len2);
	
	// copy results to host
	hipMemcpy(data.bestScores, d_bestScores, data.num_mutants * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(data.bestOffsets, d_bestOffsets, data.num_mutants * sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(d_seq1);
	hipFree(d_seq2);
	hipFree(d_bestScores);
	hipFree(d_bestOffsets);
	hipFree(d_nkArr);
	hipFree(d_ABCgrid);
}

// ========================

